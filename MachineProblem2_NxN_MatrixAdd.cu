#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <algorithm>
#include <random>
#include <ctime>

using namespace std;

#define BLOCK_WIDTH 16

// Device code
__global__ void MatrixMult_Device(const float* d_a, const float* d_b, float* d_c, const int n)
{
	int column = threadIdx.x + (blockIdx.x * blockDim.x);
	int row = threadIdx.y + (blockIdx.y * blockDim.y);
	float c_value;
	
	if ((column < n) && (row < n)) {
		c_value = 0;
		for (int k = 0; k < n; ++k) {
			c_value += d_a[(n * k) + row] * d_b[(n * column) + k];
		}
		d_c[(n * column) + row] = c_value;
	}
}

// Matrix addition - Part 2
__global__ void MatrixAddOneN(const float* d_inputMatrix, float* d_oneM,  const int n)
{
	int column = threadIdx.x + (blockIdx.x * blockDim.x);
	int row = threadIdx.y + (blockIdx.y * blockDim.y);
	float c_value;
	
	if ((column < n) && (row < n)) {
		c_value = 0;
		for (int k = 0; k < n; ++k) {
			c_value += d_inputMatrix[(n * k) + row];
		}
		d_oneM[row] = c_value;
	}
}

// Add all to one result
__global__ void MatrixAddTotal(const float* d_oneMMatrix, float* d_finalResult,  const int n)
{
	int column = threadIdx.x + (blockIdx.x * blockDim.x);
	int row = threadIdx.y + (blockIdx.y * blockDim.y);
	if ((column < n) && (row < n)) {
		*d_finalResult = 0;
		__syncthreads();
		atomicAdd(d_finalResult, d_oneMMatrix[column]);
		__syncthreads();
	}
}

// Host code
int main()
{	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    const int N = 2;
	const int arraySize = N * N;
	const int arraySizeBytes = arraySize * sizeof(float);

	hipError_t err;

	float *h_a, *h_b, *h_c, *verify_result;
	float *d_a, *d_b, *d_c;

	// Allocate space for host copies on CPU
	h_a = (float *)malloc(arraySizeBytes);
	h_b = (float *)malloc(arraySizeBytes);
	h_c = (float *)malloc(arraySizeBytes);
	verify_result = (float *)malloc(arraySizeBytes);

	// Allocate space for device copies on GPU
	hipMalloc((void **)& d_a, arraySizeBytes);
	hipMalloc((void **)& d_b, arraySizeBytes);
	hipMalloc((void **)& d_c, arraySizeBytes);

	// Fill 2D host input matrices with random single-precision floating point numbers
	int random_num_a, random_num_b;
	float range = (10.0 - (-10.0)); // from -10 to + 10
	float div = RAND_MAX / range;
	srand(time(NULL));
	
	// Populate h_A and h_B input arrays with numbers
	for(int i = 0; i < N; i++) {
		for(int j = 0; j < N; j++) {
			random_num_a = (-10) + (rand() / div); // float in range -10 to +10
			random_num_b = (-10) + (rand() / div); // float in range -10 to +10

			h_a[(j * N) + i] = random_num_a;
			h_b[(i * N) + j] = random_num_b;			
		}
	}

	
	// Calculate result on CPU
	float result;
	clock_t begin = clock();
	for(int i = 0; i < N; ++i) {
		for(int j = 0; j < N; ++j) {
			result = 0;
			for(int k = 0; k < N; ++k) {
				result += h_a[(N * k) + i] * h_b[(N * j) + k];				
			}
			verify_result[(N * j) + i] = result;
		}
	}
	clock_t end = clock();
	cout << "CPU multiplication time: " << 1000.0 * (double)(end - begin) / (double)CLOCKS_PER_SEC << endl;
	

	// Copy input matrices from host memory to device memory
	hipEventRecord(start, 0);
	hipMemcpy(d_a, h_a, arraySizeBytes, hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);

	float gpu_time = 0;
	unsigned long int counter = 0;
	while(hipEventQuery(stop) == hipErrorNotReady) {
		counter++;
	}

	hipEventElapsedTime(&gpu_time, start, stop);
	 // print the GPU times
	 printf("Time spent copying 1 NxN matrix to GPU: %.2f\n", gpu_time);

	hipMemcpy(d_b, h_b, arraySizeBytes, hipMemcpyHostToDevice);
	
	// Invoke kernel
	int NumBlocks = N / BLOCK_WIDTH;
	if (N % BLOCK_WIDTH) NumBlocks++;

	dim3 dimGrid(NumBlocks, NumBlocks, 1);
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH, 1);
	
	hipEventRecord(start, 0);
	MatrixMult_Device<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, N);
	hipEventRecord(stop, 0);

	gpu_time = 0;
	counter = 0;
	while(hipEventQuery(stop) == hipErrorNotReady) {
		counter++;
	}

	hipEventElapsedTime(&gpu_time, start, stop);
	// print the GPU times
	printf("Time spent multiplying matrices on GPU: %.2f\n", gpu_time);


	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// Copy result from device to host
	hipMemcpy(h_c, d_c, arraySizeBytes, hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	// Compare CPU results to GPU results
	bool correct = true;

	for(int i = 0; i < (N * N); i++) {
		printf("%f\n", h_c[i]);
		if(verify_result[i] != h_c[i]){
			correct = false;
			break;
		}
	}

	if(correct) {
		printf("Verification passed :)");
	}
	else {
		printf("Verification failed (:");
	}

	// Keeps terminal open until user hits 'Return' on terminal
	cin.get();


	// Machine Problem #2
	
	// Using matrix we got from previous part - h_c
	// we have h_c
	float *h_oneM;
	float *d_oneM, *d_inputMatrix;

	// Allocate space for host copies on CPU
	h_oneM = (float *)malloc(arraySizeBytes/N);

	// Allocate space for device copies on GPU
	hipMalloc((void **)& d_oneM, arraySizeBytes/N);
	hipMalloc((void **)& d_inputMatrix, arraySizeBytes);

	// Calcualte add on CPU
	result;
	for(int i = 0; i < N; ++i) {
		for(int j = 0; j < N; ++j) {
			result = 0;
			for(int k = 0; k < N; ++k) {
				result += h_c[(N * k) + i];			
			}
			h_oneM[i] = result;
		}
	}
	
	for (int i = 0; i < N; i++)
	{
		printf("%f\n", h_oneM[i]);
	}

	// Copy input matrices from host memory to device memory
	hipMemcpy(d_inputMatrix, h_c, arraySizeBytes, hipMemcpyHostToDevice);
	
	// Invoke GPU device function
	MatrixAddOneN<<<dimGrid, dimBlock>>>(d_inputMatrix, d_oneM, N);

	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// Copy result from device to host
	hipMemcpy(h_oneM, d_oneM, (arraySizeBytes/N), hipMemcpyDeviceToHost);

	// print it out
	for (int i = 0; i < N; ++i)
	{
		printf("Printing GPU result from CPU = %f\n", h_oneM[i]);
	}

	// Free device memory
	hipFree(d_oneM);
	hipFree(d_inputMatrix);

	// CPU - oneM to total value
	result = 0;
	for(int i = 0; i < N; ++i) {
		result += h_oneM[i];
	}

	printf("Final Result in CPU is %f\n", result);

	// Now lets do final result in GPU
	float *h_finalResult;
	float *d_oneMMatrix, *d_finalResult;

	// Allocate space for host copies on CPU
	h_finalResult = (float *)malloc(sizeof(float));

	// Allocate space for device copies on GPU
	hipMalloc((void **)& d_oneMMatrix, arraySizeBytes/N);
	hipMalloc((void **)& d_finalResult, sizeof(float));

	// Copy input matrices from host memory to device memory
	hipMemcpy(d_oneMMatrix, h_oneM, arraySizeBytes/N, hipMemcpyHostToDevice);

	dim3 dimBlock2(BLOCK_WIDTH, 1, 1);

	// Invoke GPU device function
	MatrixAddTotal<<<dimGrid, dimBlock2>>>(d_oneMMatrix, d_finalResult, N);

	err = hipDeviceSynchronize();
	printf("Run kernel: %s\n", hipGetErrorString(err));

	// Copy result from device to host
	hipMemcpy(h_finalResult, d_finalResult, (sizeof(float)), hipMemcpyDeviceToHost);

	printf("Final result from GPU is %f\n", *h_finalResult);

	if (*h_finalResult == result)
		printf("2nd machine problem verified.");
	else
		printf("2nd machine problem failed.");

	hipFree(d_finalResult);
	hipFree(d_oneMMatrix);
	cin.get();

    return 0;
}
